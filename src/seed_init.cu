#include "hip/hip_runtime.h"
#include <cuda_toolkit/hip/hip_vector_types.h>
#include <quadmap/device_image.cuh>
#include <quadmap/texture_memory.cuh>
#include <ctime>

namespace quadmap
{
//declear function
void generate_gradient(DeviceImage<float> &image, DeviceImage<float2> &gradient_map);
__global__ void gradient_kernel(DeviceImage<float> *image_dev_ptr, DeviceImage<float2> *gradient_dev_ptr);

//define function
void generate_gradient(DeviceImage<float> &image, DeviceImage<float2> &gradient_map)
{
	int width = gradient_map.width;      // 看来是个常数 752
	int height = gradient_map.height;    // 看来是个常数 480
    // std::cout << "frame width = " << width << ", height = " << height << std::endl;
	dim3 block;
	dim3 grid;
	block.x = 16;
	block.y = 16;
	grid.x = (width + block.x - 1) / block.x;
	grid.y = (height + block.y - 1) / block.y;
	gradient_kernel<<<grid, block>>>(image.dev_ptr, gradient_map.dev_ptr);
}
/**
 *
 * @param image_dev_ptr
 * @param gradient_dev_ptr
 * note: float2数据类型: 包含两个数据x,y
 * solution : 在opencl中利用float4格式,读取R,G两个通道的值,即CL_RG
 */
__global__ void gradient_kernel(DeviceImage<float> *image_dev_ptr, DeviceImage<float2> *gradient_dev_ptr)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;
	const int width = gradient_dev_ptr->width;
	const int height = gradient_dev_ptr->height;

	if (x >= width - 1 || y >= height - 1 || x <= 0 || y <= 0)
		return;
    // width(横)方向的梯度
	float right_color = image_dev_ptr->atXY(x+1,y);
	float left_color = image_dev_ptr->atXY(x-1,y);
	// height(纵)方向的梯度
	float down_color = image_dev_ptr->atXY(x,y+1);
	float up_color = image_dev_ptr->atXY(x,y-1);

	gradient_dev_ptr->atXY(x, y) = make_float2((right_color - left_color)/2.0, (down_color - up_color)/2.0);
}
}